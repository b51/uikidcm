#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include "yuv2rgb.cuh"

__device__ inline float clamp(float val, float mn, float mx) {
  return (val >= mn) ? ((val <= mx) ? val : mx) : mn;
}

__global__ void gpuConvertYUYVtoRGB_kernel(unsigned char* Y, unsigned char* U,
                                           unsigned char* V, unsigned char* dst,
                                           int y_stride, int u_stride,
                                           int v_stride, unsigned int width,
                                           unsigned int height) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx * 2 >= width) {
    return;
  }
  for (int i = 0; i < height; ++i) {
    int y0 = Y[i * y_stride + idx * 2 + 0];
    int y1 = Y[i * y_stride + idx * 2 + 1];
    int cr = U[i * u_stride / 2 + idx];
    int cb = V[i * v_stride / 2 + idx];

    dst[i * width * 3 + idx * 6 + 0] =
        clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128), 0.0f, 255.0f);
    dst[i * width * 3 + idx * 6 + 1] =
        clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128),
              0.0f, 255.0f);
    dst[i * width * 3 + idx * 6 + 2] =
        clamp(1.164f * (y0 - 16) + 2.018f * (cb - 128), 0.0f, 255.0f);

    dst[i * width * 3 + idx * 6 + 3] =
        clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128), 0.0f, 255.0f);
    dst[i * width * 3 + idx * 6 + 4] =
        clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128),
              0.0f, 255.0f);
    dst[i * width * 3 + idx * 6 + 5] =
        clamp(1.164f * (y1 - 16) + 2.018f * (cb - 128), 0.0f, 255.0f);
  }
}

void gpuConvertYUYVtoRGB(unsigned char* Y, unsigned char* U, unsigned char* V,
                         unsigned char* dst, int y_stride, int u_stride,
                         int v_stride, unsigned int width,
                         unsigned int height) {
  unsigned char* d_Y = NULL;
  unsigned char* d_U = NULL;
  unsigned char* d_V = NULL;
  unsigned char* d_dst = NULL;

  size_t planeSize = width * height * sizeof(unsigned char);
  size_t planeSize_Y = y_stride * height * sizeof(unsigned char);
  size_t planeSize_U = u_stride * height * sizeof(unsigned char);
  size_t planeSize_V = v_stride * height * sizeof(unsigned char);

  unsigned int flags;
  bool YIsMapped = (hipHostGetFlags(&flags, Y) == hipSuccess) &&
                   (flags & hipHostMallocMapped);
  bool UIsMapped = (hipHostGetFlags(&flags, U) == hipSuccess) &&
                   (flags & hipHostMallocMapped);
  bool VIsMapped = (hipHostGetFlags(&flags, V) == hipSuccess) &&
                   (flags & hipHostMallocMapped);
  bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) &&
                     (flags & hipHostMallocMapped);
  if (YIsMapped) {
    d_Y = Y;
    d_U = U;
    d_V = V;
    hipStreamAttachMemAsync(NULL, Y, 0, hipMemAttachGlobal);
  } else {
    hipMalloc(&d_Y, planeSize_Y);
    hipMemcpy(d_Y, Y, planeSize_Y, hipMemcpyHostToDevice);
    hipMalloc(&d_U, planeSize_U);
    hipMemcpy(d_U, U, planeSize_U, hipMemcpyHostToDevice);
    hipMalloc(&d_V, planeSize_V);
    hipMemcpy(d_V, V, planeSize_V, hipMemcpyHostToDevice);
  }
  if (dstIsMapped) {
    d_dst = dst;
    hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
  } else {
    hipMalloc(&d_dst, planeSize * 3);
  }
  unsigned int blockSize = 1024;
  unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
  gpuConvertYUYVtoRGB_kernel<<<numBlocks, blockSize>>>(
      d_Y, d_U, d_V, d_dst, y_stride, u_stride, v_stride, width, height);
  hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
  hipStreamSynchronize(NULL);
  if (!YIsMapped) {
    hipMemcpy(dst, d_dst, planeSize * 3, hipMemcpyDeviceToHost);
    hipFree(d_Y);
    hipFree(d_U);
    hipFree(d_V);
  }
  if (!dstIsMapped) {
    hipFree(d_dst);
  }
}

__global__ void gpuConvertYUYVtoRGB_kernel(e_yuyv_type type, unsigned char* src,
                                           unsigned char* dst,
                                           unsigned int width,
                                           unsigned int height) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx * 2 >= width) {
    return;
  }

  if (type == YUYV_PACKED) {
    for (int i = 0; i < height; ++i) {
      int y0 = src[i * width * 2 + idx * 4 + 0];
      int y1 = src[i * width * 2 + idx * 4 + 2];
      int cr = src[i * width * 2 + idx * 4 + 1];
      int cb = src[i * width * 2 + idx * 4 + 3];

      dst[i * width * 3 + idx * 6 + 0] =
          clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128), 0.0f, 255.0f);
      dst[i * width * 3 + idx * 6 + 1] =
          clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128),
                0.0f, 255.0f);
      dst[i * width * 3 + idx * 6 + 2] =
          clamp(1.164f * (y0 - 16) + 2.018f * (cb - 128), 0.0f, 255.0f);

      dst[i * width * 3 + idx * 6 + 3] =
          clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128), 0.0f, 255.0f);
      dst[i * width * 3 + idx * 6 + 4] =
          clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128),
                0.0f, 255.0f);
      dst[i * width * 3 + idx * 6 + 5] =
          clamp(1.164f * (y1 - 16) + 2.018f * (cb - 128), 0.0f, 255.0f);
    }
  } else if (type == YUYV_PLANNAR) {
    for (int i = 0; i < height; ++i) {
      int y0 = src[i * width + idx * 2 + 0];
      int y1 = src[i * width + idx * 2 + 1];
      int cr = src[width * height + i * width / 2 + idx];
      int cb = src[width * height + width * height / 2 + i * width / 2 + idx];

      dst[i * width * 3 + idx * 6 + 0] =
          clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128), 0.0f, 255.0f);
      dst[i * width * 3 + idx * 6 + 1] =
          clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128),
                0.0f, 255.0f);
      dst[i * width * 3 + idx * 6 + 2] =
          clamp(1.164f * (y0 - 16) + 2.018f * (cb - 128), 0.0f, 255.0f);

      dst[i * width * 3 + idx * 6 + 3] =
          clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128), 0.0f, 255.0f);
      dst[i * width * 3 + idx * 6 + 4] =
          clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128),
                0.0f, 255.0f);
      dst[i * width * 3 + idx * 6 + 5] =
          clamp(1.164f * (y1 - 16) + 2.018f * (cb - 128), 0.0f, 255.0f);
    }
  }
}

void gpuConvertYUYVtoRGB(e_yuyv_type type, unsigned char* src,
                         unsigned char* dst, unsigned int width,
                         unsigned int height) {
  unsigned char* d_src = NULL;
  unsigned char* d_dst = NULL;
  size_t planeSize = width * height * sizeof(unsigned char);

  unsigned int flags;
  bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) &&
                     (flags & hipHostMallocMapped);
  bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) &&
                     (flags & hipHostMallocMapped);

  if (srcIsMapped) {
    d_src = src;
    hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
  } else {
    hipMalloc(&d_src, planeSize * 2);
    hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
  }
  if (dstIsMapped) {
    d_dst = dst;
    hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
  } else {
    hipMalloc(&d_dst, planeSize * 3);
  }

  unsigned int blockSize = 1024;
  unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
  gpuConvertYUYVtoRGB_kernel<<<numBlocks, blockSize>>>(type, d_src, d_dst,
                                                       width, height);
  hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
  hipStreamSynchronize(NULL);

  if (!srcIsMapped) {
    hipMemcpy(dst, d_dst, planeSize * 3, hipMemcpyDeviceToHost);
    hipFree(d_src);
  }
  if (!dstIsMapped) {
    hipFree(d_dst);
  }
}
